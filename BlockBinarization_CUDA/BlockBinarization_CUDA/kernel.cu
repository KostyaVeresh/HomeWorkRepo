#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"

#include <iostream>

const int threadN = 3;
const int blockSize = threadN * threadN;
const int BLOCK_SIZE = threadN;

// �������-���� ����������� ������ ������� CUDA-�����
// ������� ������ ������������� ������� �����������
__global__ void binThreshold(
		unsigned char *scr
		, unsigned char *img
		, int width
	)
{
	__shared__ int sum;
	__shared__ int max;
	__shared__ int count;
	bool isCounted = false;
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	int tmp = width * j + i;

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		sum = 0;
	}
	__syncthreads();

	// ������� ����� ��������� � �����
	atomicAdd(&sum, scr[tmp]);
	__syncthreads();

	if (threadIdx.x == 0 && threadIdx.y == 0) {
		sum /= 255;
	}
	__syncthreads();

	// ����������� �������� ��������� �����������
	while (sum > 0) {
		if (threadIdx.x == 0 && threadIdx.y == 0) {
			max = 0;
			count = 0;
		}
		__syncthreads();

		// �� ������ �������� ������� �������
		// ������������ ������� 
		if (!isCounted) {
			atomicMax(&max, scr[tmp]);
		}
		__syncthreads();

		bool isMax = false;
		if (scr[tmp] == max) {
			atomicAdd(&count, 1);
			isCounted = true;
			isMax = true;
		}
		__syncthreads();

		if (isMax) {
			if (sum >= count) {
				atomicAdd(&sum, -1);
				img[tmp] = 255;
			} else {
				if (atomicAdd(&sum, -1) > 0) {
					img[tmp] = 255;
				}
			}
		}
		__syncthreads();
	}
}

// ����������� ������ �����
void blockProc(const cv::Mat &scr, cv::Mat &img, int x, int y)
{
	uchar block[threadN][threadN];
    uchar binBlock[threadN][threadN];

    int sum = 0;

    for (int i = 0; i < threadN; ++i) {
        const uchar *rowPtr = scr.ptr<uchar>(x + i) + y;
        for (int j = 0; j < threadN; ++j) {
            block[i][j] = rowPtr[j];
            binBlock[i][j] = 0;
            sum += rowPtr[j];
        }
    }

    sum /= 255;

    int lessThan = 256;                  // ��������� �������� ���� �����
    while (sum != 0) {
        int max = 0;
        int count = 0;

        for (int i = 0 ; i < BLOCK_SIZE; ++i) {
            for (int j = 0; j < BLOCK_SIZE; ++j) {
                if (block[i][j] < lessThan) {
                    if (max < block[i][j]) {
                        max = block[i][j];
                        count = 1;
                    } else if (max == block[i][j]) {
                        ++count;
                    }
                }
            }
        }

        lessThan = max;         // ������� ����� ������������ �������
		uchar *points[BLOCK_SIZE * BLOCK_SIZE];

        int k = 0;
        for (int i = 0; i < BLOCK_SIZE; ++i) {
            for (int j = 0 ; j < BLOCK_SIZE; ++j) {
                if (max == block[i][j]) {
                    points[k] = &binBlock[i][j];
                    ++k;
                }
            }
        }

        if (sum >= count) {
            for (int i = 0; i < count; ++i) {
                *points[i] = 255;
            }
            sum -= count;
        } else {                                    // ������������
            for (int i = 0; i < sum; ++i) {
                int pos = std::rand() % count;
                *points[pos] = 255;
                points[pos] = points[count - 1];
                --count;
            }
            sum = 0;
        }
	}

	for (int i = 0; i < BLOCK_SIZE; ++i) {        // ���������� ���� � �����������
        uchar *rowPtr = img.ptr<uchar>(x + i) + y;
        for (int j = 0; j < BLOCK_SIZE; ++j) {
            rowPtr[j] = binBlock[i][j];
        }
    }
}

int main()
{
	// �������� �����������
	cv::Mat scr = cv::imread("test\\Cameraman.png", 0);

	int rows = scr.rows + threadN - scr.rows % threadN;
    int cols = scr.cols + threadN - scr.cols % threadN;
	cv::Mat bordered = cv::Mat(rows, cols, CV_8U);
    scr.copyTo(bordered(cv::Rect(0, 0, scr.cols, scr.rows)), scr);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//----------------------  GPU ------------------------------
	hipEventRecord(start, 0);
	
	unsigned char *sourceData, *binData;

	// ��������� ������
	hipMallocManaged(&sourceData, rows * cols * sizeof(unsigned char));
	hipMallocManaged(&binData, rows * cols * sizeof(unsigned char));
	memcpy(sourceData, bordered.data, rows * cols * sizeof(unsigned char));

	int blockX = cols / threadN;
	int blockY = rows / threadN;
	dim3 cblocks(blockX, blockY);
	dim3 cthreads(threadN, threadN);

	// ������ �������-���� �� ���� �������
	binThreshold <<< cblocks, cthreads >>> (sourceData, binData, cols);
	hipDeviceSynchronize();
	cv::Mat bin(bordered.size(), CV_8U, binData);
	bin = bin(cv::Rect(0, 0, scr.cols, scr.rows));

	hipEventRecord(stop);
	float msGPU = 0.0f;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msGPU, start, stop);
    std::cout << "Elapsed GPU time: " << msGPU << " milliseconds" << std::endl;

	//----------------------- CPU --------------------------------
	hipEventRecord(start, 0);
	hipEventSynchronize(start);

	cv::Mat img1 = cv::Mat(rows, cols, CV_8U);
	cv::Mat bordered2 = cv::Mat(rows, cols, CV_8U);
	scr.copyTo(bordered2(cv::Rect(0, 0, scr.cols, scr.rows)), scr);

	// ������ ������������� �����
	//#pragma omp parallel for
	for (int i = 0; i < rows; i += threadN) {
		for (int j = 0; j < cols; j += threadN) {
            blockProc(bordered2, img1, i, j);
        }
    }
	cv::Mat img = img1(cv::Rect(0, 0, scr.cols, scr.rows));

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float msCPU = 0.0f;
	hipEventElapsedTime(&msCPU, start, stop);
    std::cout << "Elapsed CPU time: " << msCPU << " milliseconds" << std::endl;
	std::cout << std::endl << "Rate: " << (msCPU / msGPU) << " x" << std::endl;

	int screen = scr.rows > 1080 || scr.cols > 1960 ? cv::WINDOW_NORMAL : cv::WINDOW_AUTOSIZE;
	//imwrite("binarizedGPU.jpg", bin);
	//imwrite("binarizedCPU.jpg", img);

    cv::namedWindow("GPU", screen);
	cv::imshow("GPU", bin);
	cv::namedWindow("CPU", screen);
	cv::imshow("CPU", img);
	cv::waitKey();
	//getchar();

	hipFree(sourceData);
	hipFree(binData);
	hipEventDestroy(start);
	hipEventDestroy(stop);
    return 0;
}